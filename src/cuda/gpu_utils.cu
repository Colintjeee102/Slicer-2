#include "cuda/gpu_utils.h"

namespace CUDA
{
    bool check_for_gpu()
    {
        hipError_t err0;
        int nb_devices;
        err0 = hipGetDeviceCount(&nb_devices);

        if (err0 != hipSuccess || nb_devices == 0)
        {
            return false;
        }else{
            return true;
        }
    }

    int get_device_count()
    {
        int nDevices;
        hipGetDeviceCount(&nDevices);
        return nDevices;
    }

    hipDeviceProp_t get_device_properties(size_t devID)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, devID);

        return prop;
    }
}
